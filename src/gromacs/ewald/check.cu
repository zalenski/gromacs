#include <hip/hip_runtime.h>

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/real.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/timing/wallcycle.h"

#include "thread_mpi/mutex.h"

struct gpu_events {
  bool created;
  hipEvent_t event_start, event_stop;
  gpu_events() : created(false) { }
};

gpu_events gpu_events_interpol_idx;
gpu_events gpu_events_calcspline;
gpu_events gpu_events_spread;
gpu_events gpu_events_fft_r2c;
gpu_events gpu_events_solve;
gpu_events gpu_events_fft_c2r;
gpu_events gpu_events_gather;

void events_record_start(gpu_events &events) {
  if (!events.created) {
    hipEventCreate(&events.event_start);
    hipEventCreate(&events.event_stop);
    events.created = true;
  }
  hipEventRecord(events.event_start);
}

void events_record_stop(gpu_events &events, int ewcsn, int j) {
  hipEventRecord(events.event_stop);
  hipEventSynchronize(events.event_stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, events.event_start, events.event_stop);

  int idx = ewcsn - ewcsPME_INTERPOL_IDX;
  gmx_wallclock_gpu_pme.pme_time[idx][j].t += milliseconds;
  ++gmx_wallclock_gpu_pme.pme_time[idx][j].c;
}

const bool check_verbose = false;
static tMPI::mutex print_mutex;

template <typename T>
void check(const char *name, T *data, T *expected, int size, gmx_bool bDevice)
{
  print_mutex.lock();
  bool bDiff = false;
  for (int i = 0; i < size; ++i) {
    T cpu_v = expected[i];
    T gpu_v;
    if (bDevice) {
      hipMemcpy(&gpu_v, &data[i], sizeof(T), hipMemcpyDeviceToHost);
    } else {
      gpu_v = data[i];
    }
    T diff = gpu_v - cpu_v;
    if (check_verbose) {
      fprintf(stderr, " %d:%f(%f)", i, (double) cpu_v, (double) diff);
    }
    if (diff != 0) {
      if (!bDiff) {
	fprintf(stderr, "%s\n", name);
	bDiff = true;
      }
      T absdiff = diff > 0 ? diff : -diff;
      T abscpu_v = cpu_v > 0 ? cpu_v : -cpu_v;
      T reldiff = absdiff / (abscpu_v > 1e-11 ? abscpu_v : 1e-11);
      if (reldiff > .000001) {
	fprintf(stderr, "%.0fppm", (double) (reldiff * 1e6));
	if (reldiff > .0001) {
	  fprintf(stderr, " value %f vs %f ", (double) cpu_v, (double) gpu_v);
	}
      } else {
	fprintf(stderr, "~");
      }
    }
  }
  if (bDiff) {
    fprintf(stderr, "\n");
  }
  print_mutex.unlock();
}

void check_int(const char *name, int *data, int *expected, int size, gmx_bool bDevice)
{
  check(name, data, expected, size, bDevice);
}

void check_real(const char *name, real *data, real *expected, int size, gmx_bool bDevice)
{
  check(name, data, expected, size, bDevice);
}

void print_lock() {
  print_mutex.lock();
}

void print_unlock() {
  print_mutex.lock();
}
